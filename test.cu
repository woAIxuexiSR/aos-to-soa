#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>

#include "test.h"

using namespace std;

__global__ void kernel(SOA<Interval> soa, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;
    // set all the values
    // soa[i] = { sqrtf(i + 1), sqrtf(i + 2) };
    // set only the low value
    soa.low[i] = sqrtf(i + 1);
}

__global__ void show(SOA<Interval> soa, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;
    // get all the values
    // auto val = Interval(soa[i]);
    // printf("low: %f, high: %f\n", val.low, val.high);
    // get only the low value
    printf("low: %f\n", soa.low[i]);
}

int main()
{
    constexpr int N = 1 << 20;
    constexpr int block_size = 256;

    SOA<Interval> soa(N);
    kernel<<<(N + block_size - 1) / block_size, block_size>>>(soa, N);
    hipDeviceSynchronize();

    show<<<10, 1>>>(soa, N);
    hipDeviceSynchronize();

    return 0;
}